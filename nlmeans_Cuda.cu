

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <time.h>

struct timeval tstart;
struct timeval tic();
double toc(struct timeval begin);
double*  addNoise(double* Im, int imSize, double scalar);
double   GaussianNoise(double sigma, double x);
double** createMatrix(int row, int col);
double** readFile(char* filename, int rowSize, int colSize);
void     mywriteFile(double**A, char* filename, int rowSize, int colSize);
double** oneDim2twoDim(double* A, int len);
double*  twoDim2oneDim(double** A, int len);
void     printArray(double **A, double *B, int len, int dim);
double*  GaussianKernel(int krnl_sz, double sigma);


//NUM_THREADS SHOULD BE IMAGE_SIZE^2
__global__ void NonLocalMeans(double* Im, int imSize, double imSigma, double* patch, 
                                          int ptSize, double* ptW, double* W, double* If){
    
    double x, x2, tmp, D = 0, normZ=0;
    int i2 = 0, j3 = 0,
        size = pow( imSize, 2 ),
        len  = pow( ptSize, 2 ),
        id   = threadIdx.x + blockDim.x * blockIdx.x;

    for(int i = 0; i < size; i++)  
    {
        for(int j=i*len, j2=0; j < i*len+len; j++, j2++)  /* This loop checks every time, a different patch from the patches-list */
        {  
            if(j3 == ptSize){
                i2 ++;
                j3 = 0;
            }
            if(i2 == ptSize) i2 = 0;
    
            x  = patch[id*len+j2];                   
            x2 = patch[j];
            if(x != 0 && x2 != 0)
            {   
                tmp = pow( (x-x2), 2 );         
                tmp *= ptW[i2*ptSize+j3];   
                D   += tmp;  
            }
            j3++;          
        }

        D = exp( (-D) / pow(imSigma, 2) );
        W[id] = D;
        If[id] += W[id] * Im[i];
        normZ += D;
        D = 0;
    }

    If[id] /= normZ; 
   
}


/* Returns all the patches layed into a vector */
__global__ void findPatches(double* Im, double* patch, int imageSize,
                                        int patchSize){
    
    int size = pow( imageSize, 2 ) * pow( patchSize, 2 );
    int r, r2, c, c2, cnt = 0, range = (patchSize - 1) / 2;

    for(int i = 0; i < imageSize; i++)
        for(int j = 0; j < imageSize; j++)
        {   
            for(r=i-range, r2=0; r2 < patchSize; r++, r2++)
                for(c=j-range, c2=0; c2 < patchSize; c++, c2++)
                { 
                    if((r >= imageSize) || (c >= imageSize)) patch[cnt++] = 0;
                    else if(r < 0  || c < 0) patch[cnt++] = 0;
                    else if(r >= 0 || c >= 0) patch[cnt++] = Im[r*imageSize+c];
                }
        }
}



int main(){
    
    int    pSize   = 7,
           iSize   = 128,
           size    = pow(iSize, 2),
           nblocks = size / 512;
    double pSigma  = 0.8,
           iSigma  = 0.08,
           nEffect = 0.5;
    
    double** nIm  = readFile("im128.txt", iSize, iSize);

    double* Im    = twoDim2oneDim(nIm, iSize);
    double* noise = addNoise(Im, size, nEffect);
    double* krnl  = GaussianKernel(pSize, pSigma);
    double* If    = (double *)malloc(size * sizeof(double));

    double *dnoise, *dkrnl, *patch, *W, *dfilt;
    int size1 = pow( iSize, 2 ) * sizeof(double),
        size2 = pow( pSize, 2 ) * sizeof(double),
        size3 = pow( iSize, 2 ) * pow( pSize, 2 ) * sizeof(double);

    hipMalloc((void **)&dnoise, size1);
    hipMalloc((void **)&dfilt , size1);
    hipMalloc((void **)&dkrnl , size2);
    hipMalloc((void **)&patch , size3);
    hipMalloc((void **)&W     , size1);
    
    hipMemcpy(dnoise, noise, size1, hipMemcpyHostToDevice);
    hipMemcpy(dkrnl, krnl, size2, hipMemcpyHostToDevice);
    
    tstart = tic();

    findPatches<<<1,1>>>(dnoise, patch, iSize, pSize);
    NonLocalMeans<<<nblocks,512>>>(dnoise, iSize, iSigma, patch, pSize, dkrnl, W, dfilt);
    hipDeviceSynchronize();
    
    double duration = toc(tstart);
    printf("~ Duration: %f sec\n", duration);

    hipMemcpy(If, dfilt, size1, hipMemcpyDeviceToHost);

    double** If2 = oneDim2twoDim(If, iSize);
    mywriteFile(If2, "denoise.txt", iSize, iSize);

}


/* Calculates spacial-gaussian weight */
double* GaussianKernel(int krnl_sz, double sigma){

    double *W = (double *)malloc(pow( krnl_sz, 2 ) * sizeof(double)),
             x, y, d, sum = 0.0,
             c = 2 * pow( sigma, 2 );

    for(int i = 0; i < krnl_sz; i++)
        for(int j = 0; j < krnl_sz; j++)
        {   
            x = i - (krnl_sz - 1) / 2.0;
            y = j - (krnl_sz - 1) / 2.0;
            d = x * x + y * y;
            W[i*krnl_sz+j] = exp( -(d) / c ) / (M_PI * c);
            sum += W[i*krnl_sz+j];
        }

    double max[krnl_sz];
    int i = 0;

    for(i = 0, max[i] = 0; i < krnl_sz; i++)
        for(int j = 0; j < krnl_sz; j++)
        {
            W[i*krnl_sz+j] /= sum;
            if(j==0) max[i] = W[i*krnl_sz+j];
            else if(W[i*krnl_sz+j] > max[i]) 
                max[i] = W[i*krnl_sz+j]; 
        }

    for(int i = 0; i < krnl_sz; i++)
        for(int j = 0; j < krnl_sz; j++)
            W[i*krnl_sz+j] /= max[i];

    return W;
}



void printArray(double **A, double *B, int len, int dim){

    if(dim == 2)
    {
        for(int i = 0; i < len; i++){
            for(int j = 0; j < len; j++){
                printf("%f ", A[i][j]);
            }
            printf("\n");
        }
    }
    else{
        for(int i = 0; i < len; i++) printf("%f\n", B[i]);
    }
}


double** readFile(char* filename, int rowSize, int colSize){

    double** A = (double **)malloc(rowSize * sizeof(double));
    for(int i=0; i<colSize; i++) A[i] = (double *)malloc(colSize * sizeof(double));

    FILE *fp = fopen(filename, "r");

    for(int i = 0; i < rowSize; i++)
        for(int j = 0; j < colSize; j++)
        {
            fscanf(fp, "%lf %*c", &A[i][j]);
        }

    return A;
}


void mywriteFile(double**A, char* filename, int rowSize, int colSize){

    FILE *fp = fopen(filename, "w");

    for(int i = 0; i < rowSize; i++){
        for(int j = 0; j < colSize; j++)
        {
            fprintf(fp, "%lf,", A[i][j]);
        }
        fprintf(fp, "\n");
    }
}


double GaussianNoise(double sigma, double x){

    return (1 / (sigma*sqrt(2*M_PI)))*exp((-x*x) / (2*sigma*sigma));
}


double* addNoise(double* Im, int imSize, double scalar){

    double *noise = (double *)malloc(imSize * sizeof(double)),
            value, effect;

    for(int i = 0; i < imSize; i++)
    {
        value    = ((double)( rand() ) / RAND_MAX*20 - 10);
        effect   = GaussianNoise(2, value) - 0.1;
        noise[i] = (scalar * effect + 1) * Im[i];
    }
    
    return noise;
}


/* 2D to 1D */
double* twoDim2oneDim(double** A, int len){

    double* C = (double *)malloc(len * len * sizeof(double ));
    int cnt = 0;

    for(int i = 0; i < len; i++)
        for(int j = 0; j < len; j++)
            C[cnt++] = A[i][j];

    return C;
}


/* 1D to 2D */
double** oneDim2twoDim(double* A, int len){

    double** D = (double **)malloc(len * sizeof(double*));
    for(int f = 0; f < len; f++) D[f] = (double *)malloc(len * sizeof(double));
    int cnt = 0;

    for(int i = 0; i < len; i++)
        for(int j = 0; j < len; j++)
            D[i][j] = A[cnt++];

    return D;
}



/* 2-D random matrix */
double** createMatrix(int row, int col){

    srand(time( NULL ));
    double** mat = (double **)malloc(row * sizeof(double *));
    for(int i=0; i<row; i++) mat[i] = (double *)malloc(col * sizeof(double));

    for(int i=0; i<row; i++)
        for(int j=0; j<col; j++)
            mat[i][j] = (double)( rand() ) / (double)( RAND_MAX );

    return mat;
}


struct timeval tic(){
    
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv;
}

double toc(struct timeval begin){
    
    struct timeval end;
    gettimeofday(&end, NULL);
    double stime = ((double)(end.tv_sec-begin.tv_sec)*1000)+
                            ((double)(end.tv_usec-begin.tv_usec)/1000);
    stime /= 1000;
    return stime;
}


